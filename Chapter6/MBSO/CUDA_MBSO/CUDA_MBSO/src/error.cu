/*	
 *	Copyright (C) 2011, Federico Raimondo (fraimondo@dc.uba.ar)
 *	
 *	This file is part of Cudaica.
 *
 *  Cudaica is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  any later version.
 *
 *  Cudaica is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 * 
 *  You should have received a copy of the GNU General Public License
 *  along with Cudaica.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "../include/error.h"

/*
 * Reset the errors waiting for being fetched.
 */ 



/*
 * Handles an error
 */ 
void HandleCudaError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR::%s (%x) in %s at line %d\n", hipGetErrorString( err ), err,
                file, line );
        hipError_t newerr = hipGetLastError();
        if (newerr != err && newerr != hipSuccess) {
		}
        exit( EXIT_FAILURE );
    }
}
