#include "hip/hip_runtime.h"
#include "../include/config.h"
#include ""
__global__ void global_setupRandomState(hiprandState * states, natural seed)
{
	int var_random = blockDim.x * blockDim.y * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x + threadIdx.y * blockDim.x;
	hiprand_init(seed, var_random, 0, &states[var_random]);
}
extern "C"
void API_setupRandomState(dim3 blocks, dim3 threads, hiprandState * states, natural seed)
{
	global_setupRandomState << <blocks, threads >> >(states, seed);
}


